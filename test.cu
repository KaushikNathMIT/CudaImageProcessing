#include "hip/hip_runtime.h"



#include <stdio.h>

#define M 6

__global__ void add (int *A, int *B, int *C) {
    int idx = blockIdx.x;
    printf("idx = %d\n", idx);
    C[idx] = A[idx] + B[idx];
}

int main () {

    // Host copies of the variables
    int A[M], B[M], C[M];

    int i, j;
    for (i = 0; i < M; ++i) {
        A[i] = i + 1;
        B[i] = M - i - 1;
    }

    // Device copies of the variables
    int *d_a, *d_b, *d_c;

    int size = sizeof(int) * M;

    // Allocate memories to device copies of the objects
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &B, size, hipMemcpyHostToDevice);

    // Launch kernel onto the device
    add<<<M, 1>>>(d_a, d_b, d_c);

    // Copy the result back to the host
    hipMemcpy(&C, d_c, size, hipMemcpyDeviceToHost);

    // Outpoooot it
    printf("A:\n");
    for (j = 0; j < M; ++j) {
        printf("%d\t", A[j]);
    }
    printf("\n");

    printf("B:\n");
    for (j = 0; j < M; ++j) {
        printf("%d\t", B[j]);
    }
    printf("\n");

    printf("A + B:\n");
    for (j = 0; j < M; ++j) {
        printf("%d\t", C[j]);
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    getchar();

    return 0;
}